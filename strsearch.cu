#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include<iostream>
using namespace std;

// GPU method
__global__ void searchText(char* data, char* keyword, int dataLen,int keyLen)
{
	int found = 0;
	int i = (blockIdx.x * 1024) + threadIdx.x;
				int checkNext = i;
				for(int j = 0;j < keyLen; j++)
                        	{
					if(data[checkNext] == keyword[j])
                        		{
						checkNext++;
						if(j==keyLen - 1)
						{
							found = 1;
							break;
						}
					
                        		}
                        		else
                        		{
						break;
                        		}
				}	
	
	if(found == 1){
		printf("Match found at %d\n",i);	
		
	}
}


int main(int argc, char* argv[])
{
	char* str = (char*)malloc(512 * sizeof(char));
	char* str1 = (char*)malloc(512 * sizeof(char));
	printf("Enter the input file which has to be searched\n");
	scanf ("%s",str);
	printf("Enter the dictionary with the keywords to be searched \n");
	scanf("%s",str1);
	printf("input = %s\ndict = %s\n",str,str1);

	
	char *buf,*tok;
	buf = (char*)malloc(500 *sizeof(char));
	tok = (char*)malloc(500 *sizeof(char));
	FILE *f = fopen(str, "r");
	fseek(f, 0, SEEK_END);
	long fsize = ftell(f);
	fseek(f, 0, SEEK_SET);

	char *text = (char *)malloc((fsize + 1) * sizeof(char));
	printf("reading..\n");
	fread(text, fsize, 1, f);
	printf("done...\n");
	fclose(f);
	int noOfBlocks = strlen(text)/1024;	
	noOfBlocks++;
	printf("text size = %d\nfsize = %d\n",noOfBlocks,fsize);
	

	char* d_text;
	hipMalloc((void**)&d_text, strlen(text) * sizeof(char));
	hipMemcpy(d_text, text, strlen(text) * sizeof(char), hipMemcpyHostToDevice);
	FILE *f1 = fopen(str1,"r");
        while(fgets(buf,512,f1))
	{
	
	char* keys = (char*)malloc(128 * sizeof(char));
        tok = strtok(buf,"\t"); 
        printf("searching for = %s\n",tok);
	strcpy(keys,tok);
        hipSetDevice(0);

        char* d_keys;
       
        hipMalloc((void**)&d_keys, strlen(keys) * sizeof(char));
        hipMemcpy(d_keys, keys, strlen(keys) * sizeof(char), hipMemcpyHostToDevice);
       
        searchText<<<noOfBlocks, 1024>>>(d_text, d_keys, strlen(text),strlen(keys));

        hipDeviceSynchronize();

        hipFree(d_keys);

        free(keys);
	}
	hipFree(d_text);
	free(text);
	
	
        return 0;
}
